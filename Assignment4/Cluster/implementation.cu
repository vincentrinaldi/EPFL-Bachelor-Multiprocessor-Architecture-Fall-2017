#include "hip/hip_runtime.h"
/*
============================================================================
Filename    : algorithm.c
Author      : Vincent Rinaldi
SCIPER      : 239759
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;


// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}


// CUDA Kernel function
__global__ void kernel(double* input, double* output, int length) {

    int i   =	threadIdx.x + blockIdx.x * blockDim.x;
    int j   =	threadIdx.y + blockIdx.y * blockDim.y;

    int midSquare = (i == length/2 - 1 && j == length/2 - 1) || (i == length/2 && j == length/2 - 1) || (i == length/2 - 1 && j == length/2) || (i == length/2 && j == length/2);

    if ((i > 0) && (i < length-1) && (j > 0) && (j < length-1) && (!midSquare)) {
	output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
				    input[(i-1)*(length)+(j)]   +
				    input[(i-1)*(length)+(j+1)] +
				    input[(i)*(length)+(j-1)]   +
				    input[(i)*(length)+(j)]     +
				    input[(i)*(length)+(j+1)]   +
				    input[(i+1)*(length)+(j-1)] +
				    input[(i+1)*(length)+(j)]   +
				    input[(i+1)*(length)+(j+1)] ) / 9;
    }
							
}


// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);
    

    /* Preprocessing goes here */
    
    // declare device arrays
    double *input_d;
    double *output_d;
    
    // set device to be used for GPU executions
    hipSetDevice(0);
    
    // set number of threads per block and number of blocks in the grid used in a kernel invocation 
    int numThreadsBlock = 8;
    int numBlocksGrid = (length % numThreadsBlock != 0) ? (length / numThreadsBlock + 1) : (length / numThreadsBlock);
    
    // makes coordinates of blocks and threads indexes to work in 2 dimensions
    dim3 numThreadsPerBlock(numThreadsBlock, numThreadsBlock);
    dim3 numBlocksInGrid(numBlocksGrid, numBlocksGrid);
    
    // allocate arrays on device 
    if (hipMalloc((void **) &input_d, length*length*sizeof(double)) != hipSuccess)
	cout << "error in hipMalloc" << endl;
    if (hipMalloc((void **) &output_d, length*length*sizeof(double)) != hipSuccess)
	cout << "error in hipMalloc" << endl;
		

    // copy from host to device step
    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */   
    if (hipMemcpy(input_d, input, length*length*sizeof(double), hipMemcpyHostToDevice) != hipSuccess)
	cout << "error in hipMemcpy" << endl;
    if (hipMemcpy(output_d, output, length*length*sizeof(double), hipMemcpyHostToDevice) != hipSuccess)
	cout << "error in hipMemcpy" << endl;   
    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);
    
    
    // GPU calculation step
    hipEventRecord(comp_start);
    /* GPU calculation goes here */
    double *temp_d;
    for (int i = 0; i < iterations; i++) {
	kernel<<<numBlocksInGrid, numThreadsPerBlock>>>(input_d, output_d, length);
	if (i != iterations-1) {
	    temp_d = input_d;
	    input_d = output_d;
	    output_d = temp_d;
	}
    }
    hipDeviceSynchronize();
    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);
    

    // copy from device to host step
    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */  
    if (hipMemcpy(output, output_d, length*length*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess)
	cout << "error in hipMemcpy" << endl;
    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);
    

    /* Postprocessing goes here */
    
    // cleanup
    hipFree(input_d);
    hipFree(output_d);
    

    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}
